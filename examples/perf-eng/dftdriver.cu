#include "hip/hip_runtime.h"
//  #include "mddft3d.cu"		## File to include is defined on command line with --pre-include
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

//  Size will be defined when compiling 
//  #define M		100
//  #define N		224
//  #define K		224
//  #define FUNCNAME		mddft3d

int main() {

//	int M, N, K;
//	M=80;
//	N=80;
//	K=80;

	double *X, *Y;

	hipMalloc(&X,M*N*K*2*sizeof(double));
	hipMalloc(&Y,M*N*K*2*sizeof(double));

	double *host_X = new double[M*N*K*2];
	for (int m = 0; m < M; m++) {
		for (int n = 0; n < N; n++) {
			for (int k = 0; k < K; k++) {
				host_X[(k + n*K + m*N*K)*2 + 0] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
				host_X[(k + n*K + m*N*K)*2 + 1] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
			}
		}
	}

	hipMemcpy(X, host_X, M*N*K*2*sizeof(double), hipMemcpyHostToDevice);

	hipEvent_t start, stop, custart, custop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&custart);
	hipEventCreate(&custop);

//	hipEventRecord(start);
//	FUNCNAME(Y, X);
//	hipEventRecord(stop);

	checkCudaErrors ( hipEventRecord(start) );
	int iters = 100;
	for ( int ii = 0; ii < iters; ii++ ) {
		FUNCNAME(Y, X);
		checkCudaErrors ( hipGetLastError () );
	}
	checkCudaErrors( hipEventRecord(stop) );
	hipEventSynchronize(stop);
	hipDeviceSynchronize();

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipfftDoubleComplex *cufft_Y; 
	hipMalloc(&cufft_Y, M*N*K * sizeof(hipfftDoubleComplex));

	hipfftHandle plan;
	if (hipfftPlan3d(&plan, M, N, K,  HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
		exit(-1);
	}
 
	hipEventRecord(custart);
	for ( int ii = 0; ii < iters; ii++ ) {
		if (hipfftExecZ2Z(
				plan,
				(hipfftDoubleComplex *) X,
				(hipfftDoubleComplex *) cufft_Y,
				HIPFFT_BACKWARD
				) != HIPFFT_SUCCESS) {
			printf("hipfftExecZ2Z launch failed\n");
			exit(-1);
		}
	}
	hipEventRecord(custop);
	hipEventSynchronize(custop);

	float cumilliseconds = 0;
	hipEventElapsedTime(&cumilliseconds, custart, custop);
 
	hipDeviceSynchronize();


	printf("%f\tms (SPIRAL) vs\t%f\tms (cufft), averaged over %d iterations ##PICKME## \n",
		   milliseconds / iters, cumilliseconds / iters, iters);
 
	if (hipGetLastError() != hipSuccess) {
		printf("hipfftExecZ2Z failed\n");
		exit(-1);
	}
 
	hipfftDoubleComplex *host_Y       = new hipfftDoubleComplex[M*N*K];
	hipfftDoubleComplex *host_cufft_Y = new hipfftDoubleComplex[M*N*K];

	hipMemcpy(host_Y      ,       Y, M*N*K*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	hipMemcpy(host_cufft_Y, cufft_Y, M*N*K*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

	bool correct = true;

	for (int m = 0; m < 1; m++) {
		for (int n = 0; n < N; n++) {
			for (int k = 0; k < K; k++) {
				hipfftDoubleComplex s = host_Y      [k + n*K + m*N*K];
				hipfftDoubleComplex c = host_cufft_Y[k + n*K + m*N*K];
	    
				bool elem_correct =
					(abs(s.x - c.x) < 1e-7) &&
					(abs(s.y - c.y) < 1e-7);
				correct &= elem_correct;
				if (!elem_correct) 
				{
					correct = false;
					// printf("error at (%d,%d,%d): %f+%fi instead of %f+%fi\n", k, n, m, s.x, s.y, c.x, c.y);
				}
			}
		}
	}
end_check:
	printf("Correct: %s\t\t##PICKME## \n", (correct ? "True" : "False") );
}
