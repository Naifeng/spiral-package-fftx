#include "hip/hip_runtime.h"
//  #include "mddft3d.cu"		## File to include is defined on command line with --pre-include
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

//  Size will be defined when compiling 
//  #define M		100
//  #define N		224
//  #define K		224
//  #define FUNCNAME		mddft3d

static void buildInputBuffer(double *host_X, double *X)
{
	for (int m = 0; m < M; m++) {
		for (int n = 0; n < N; n++) {
			for (int k = 0; k < K; k++) {
				host_X[(k + n*K + m*N*K)*2 + 0] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
				host_X[(k + n*K + m*N*K)*2 + 1] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
			}
		}
	}

	hipMemcpy(X, host_X, M*N*K*2*sizeof(double), hipMemcpyHostToDevice);
	return;
}

int main() {

//	int M, N, K;
//	M=80;
//	N=80;
//	K=80;

	double *X, *Y;

	hipMalloc(&X,M*N*K*2*sizeof(double));
	hipMalloc(&Y,M*N*K*2*sizeof(double));

	double *host_X = new double[M*N*K*2];

	hipEvent_t start, stop, custart, custop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&custart);
	hipEventCreate(&custop);

//	hipEventRecord(start);
//	FUNCNAME(Y, X);
//	hipEventRecord(stop);

	#ifndef N_ITERS
	#define N_ITERS 20
	#endif
	
//	checkCudaErrors ( hipEventRecord(start) );
	int iters = N_ITERS;		// = 100;  // use smaller number due to overhead of initializing buffers
	float milliseconds[N_ITERS];
	float cumilliseconds[N_ITERS];

	hipfftDoubleComplex *cufft_Y; 
	hipMalloc(&cufft_Y, M*N*K * sizeof(hipfftDoubleComplex));

	hipfftHandle plan;
	if (hipfftPlan3d(&plan, M, N, K,  HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
		exit(-1);
	}

	for ( int ii = 0; ii < iters; ii++ ) {
		init_mddft3d();
		checkCudaErrors ( hipGetLastError () );
 
		// set up data in input buffer
		buildInputBuffer(host_X, X);

		checkCudaErrors ( hipEventRecord(start) );
		FUNCNAME(Y, X);
		checkCudaErrors ( hipGetLastError () );
		checkCudaErrors( hipEventRecord(stop) );
		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds[ii], start, stop);
		destroy_mddft3d();
		checkCudaErrors ( hipGetLastError () );

		hipEventRecord(custart);
		if (hipfftExecZ2Z(
				plan,
				(hipfftDoubleComplex *) X,
				(hipfftDoubleComplex *) cufft_Y,
				HIPFFT_BACKWARD
				) != HIPFFT_SUCCESS) {
			printf("hipfftExecZ2Z launch failed\n");
			exit(-1);
		}
		hipEventRecord(custop);
		hipEventSynchronize(custop);

		hipEventElapsedTime(&cumilliseconds[ii], custart, custop);

	}

	hipDeviceSynchronize();

	if (hipGetLastError() != hipSuccess) {
		printf("hipfftExecZ2Z failed\n");
		exit(-1);
	}

	printf("cube = [ %d, %d, %d ]\t\t ##PICKME## \n", M, N, K);
	for ( int ii = 0; ii < iters; ii++ ) { 
		printf("%f\tms (SPIRAL) vs\t%f\tms (cufft), iterations [%d] ##PICKME## \n",
		   milliseconds[ii], cumilliseconds[ii], ii);
	} 
 
	hipfftDoubleComplex *host_Y       = new hipfftDoubleComplex[M*N*K];
	hipfftDoubleComplex *host_cufft_Y = new hipfftDoubleComplex[M*N*K];

	hipMemcpy(host_Y      ,       Y, M*N*K*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	hipMemcpy(host_cufft_Y, cufft_Y, M*N*K*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

	bool correct = true;
	int errCount = 0;

	for (int m = 0; m < 1; m++) {
		for (int n = 0; n < N; n++) {
			for (int k = 0; k < K; k++) {
				hipfftDoubleComplex s = host_Y      [k + n*K + m*N*K];
				hipfftDoubleComplex c = host_cufft_Y[k + n*K + m*N*K];
	    
				bool elem_correct =
					(abs(s.x - c.x) < 1e-7) &&
					(abs(s.y - c.y) < 1e-7);
				correct &= elem_correct;
				if (!elem_correct && errCount < 10) 
				{
					correct = false;
					errCount++;
					//  printf("error at (%d,%d,%d): %f+%fi instead of %f+%fi\n", k, n, m, s.x, s.y, c.x, c.y);
				}
			}
		}
	}

end_check:
	printf("Correct: %s\t\t##PICKME## \n", (correct ? "True" : "False") );
}
